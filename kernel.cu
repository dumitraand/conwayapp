#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include <>


__global__ void cellsKernel(char *cells, int height, int width, char *resultCells,
			    char *borderTop, char *borderRight, char *bordertBot, char *borderLeft)
{
	int worldSize = height * width;
	int currentCellX, currentCellY, aliveCells, currentRow;

	int N, NE, E, SE, S, SW, W, NW;

	for (int cellId = blockIdx.x * blockDim.x + threadIdx.x; cellId < worldSize; cellId += blockDim.x * gridDim.x) {
		currentCellY = cellId % width; // cell's index from the **matrix**
		currentCellX = cellId - currentCellY; // the number of cells in the **matrix** until the current one
		currentRow = cellId / width;

		aliveCells = 0;

		N = (currentRow == 0) ? borderTop[currentCellY] : cells[currentCellX - width + currentCellY];
		S = (currentRow + 1 == height) ? bordertBot[currentCellY] : cells[currentCellX + width + currentCellY];
		W = (currentCellY == 0) ? borderLeft[currentRow + 1] : cells[currentCellX + currentCellY - 1];
		E = (currentCellY + 1 == width) ? borderRight[currentRow] : cells[currentCellX + currentCellY + 1];

		if (currentRow == 0)
			NE = borderTop[currentCellY + 1];
		else if (currentCellY + 1 == width)
			NE = borderRight[currentRow - 1];
		else
			NE = cells[currentCellX - width + currentCellY + 1];
		if (currentCellY == 0)
			NW = borderLeft[currentCellY];
		else if (currentRow == 0)
			NW = borderTop[currentCellY - 1];
		else
			NW = cells[currentCellX - width + currentCellY - 1];
		if (currentRow + 1 == height)
			SE = bordertBot[currentCellY + 1];
		else if (currentCellY + 1 == width)
			SE = borderRight[currentRow + 1];
		else
			SE = cells[currentCellX + width + currentCellY + 1];
		if (currentCellY == 0)
			SW = borderLeft[currentRow + 2];
		else if (currentRow + 1 == height)
			SW = bordertBot[currentCellY - 1];
		else
			SW = cells[currentCellX + width + currentCellY - 1];

		aliveCells = N + S + E + W + NE + SE + SW + NW;

		resultCells[currentCellX + currentCellY] = (aliveCells == 3 || (aliveCells == 2 && cells[currentCellX + currentCellY] == 1)) ? 1 : 0;
	}
}

void computeCells(char *&cells, int height, int width, char *&resultCells, char threadsCount,
		  char *borderTop, char *borderRight, char *borderBot, char *borderLeft)
{
	if ((width * height) % threadsCount != 0) {
		fprintf(stderr, "%s", "The product of square dimensions must be multiple of the number of threads!\n");
		printf("%s", "The product of square dimensions must be multiple of the number of threads!\n");
		exit(1);
	}

	int blocksCount = min(32768, (height * width) / threadsCount);

	cellsKernel <<<blocksCount, threadsCount >>> (cells, height, width, resultCells, borderTop, borderRight, borderBot, borderLeft);
}

int getGreatestDivisor(int n)
{
	int res = n;

	for (int i = 2; i <= sqrt(n); i++) {
		while (res % i == 0) {
			if (res <= 1024)
				return res;
			res /= i;
		}
		if (res <= 1024)
			return res;
	}

	return res;
}

extern "C"  char* newGeneration(char *h_cells, char *h_borderTop, char *h_borderBot,
		   char *h_borderRight, char *h_borderLeft, int height, int width)
{
	char *d_cells, *d_resultCells, *d_borderTop, *d_borderRight, *d_borderBot, *d_borderLeft;

	int worldSize = height * width;
	int num_threads = height * width;

	hipMalloc(&d_cells, worldSize * sizeof(char));

	hipMalloc(&d_resultCells, worldSize * sizeof(char));

	hipMalloc(&d_borderTop, (width + 1) * sizeof(char));

	hipMalloc(&d_borderRight, (height) * sizeof(char));

	hipMalloc(&d_borderBot, (width + 1) * sizeof(char));

	hipMalloc(&d_borderLeft, (height + 2) * sizeof(char));

	hipMemcpy(d_cells, h_cells, worldSize * sizeof(char), hipMemcpyHostToDevice);

	hipMemcpy(d_borderTop, h_borderTop, width + 1, hipMemcpyHostToDevice);

	hipMemcpy(d_borderBot, h_borderBot, width + 1, hipMemcpyHostToDevice);

	hipMemcpy(d_borderRight, h_borderRight, height, hipMemcpyHostToDevice);

	hipMemcpy(d_borderLeft, h_borderLeft, height + 2, hipMemcpyHostToDevice);

	num_threads = min(getGreatestDivisor(num_threads), width);

	computeCells(d_cells, height, width, d_resultCells, num_threads, d_borderTop,
		     d_borderRight, d_borderBot, d_borderLeft);

	hipMemcpy(h_cells, d_resultCells, worldSize * sizeof(char), hipMemcpyDeviceToHost);

	hipFree(d_cells);
	hipFree(d_resultCells);
	hipFree(d_borderBot);
	hipFree(d_borderLeft);
	hipFree(d_borderRight);
	hipFree(d_borderTop);

	return h_cells;
}
